#include "hip/hip_runtime.h"
#include "InitCuda.h"
#include <hip/hip_runtime.h>

// dummy CUDA kernel
__global__ void dummyKernel() {
    // do nothing
}

void initCuda() {
    dummyKernel<<<1,1>>>();
    hipDeviceSynchronize();
}
