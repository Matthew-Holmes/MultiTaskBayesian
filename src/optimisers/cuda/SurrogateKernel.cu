#include "hip/hip_runtime.h"
#include "SurrogateKernel.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__device__ __constant__ float sg_d;
__device__ __constant__ float l_d;
__device__ __constant__ float a_d;

// CUDA kernel
__global__ void innerEvaluationsKernel(
    float* V, int Vstride,
    float* D, int Dstride,
    float* W,
    float* muPred,
    float* sgPred,
    float* innerMerit,
    const float* S,
    const float* yDiff,
    const float* K,
    const float* lb, const float* ub,
    int ni) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i >= ni) { return; }

    // map the random values into the bounds 
    // since currently in the range [0,1]
    int vi = i * Vstride; // offset in concatenated sample space vectors
    for (int j = 0; j != Vstride; j++) {
        V[vi + j] = lb[j] + (ub[j] - lb[j]) * V[vi + j];
    } 

    // fill Dstride distances using S and kernel params
    int di = i * Dstride; // offset in concatenated weight space vectors
    for (int j = 0; j != Dstride; ++j) {
        // compute Euclidean distance for sample j
        float dj = 0.0; 
        for (int k = 0; k != Vstride; ++k) {
            float djk =  S[j * Vstride + k] - V[vi + k];
            dj += djk * djk;
        }        
        dj = sqrtf(dj); // euclidean distance
        dj = sg_d * sg_d * expf( (-1.0 * dj) / (2.0 * l_d * l_d));
        D[di + j] = dj;
    }

    // compute Dstride weight values, matrix vector multiplication
    for (int j = 0; j != Dstride; ++j) {
        float wd = 0.0;
        for (int k = 0; k != Dstride; ++k) {
            wd += K[j*Dstride + k] * D[di + k];
        }
        W[di + j] = wd;
    }

    // compute muPred
    float mu_i = 0.0;
    for (int j = 0; j != Dstride; ++j) {
        mu_i += W[di + j] * W[di + j];    
    }
    muPred[i] = mu_i;
    
    // compute sgPred
    float dot_i= 0.0;
    for (int j = 0; j != Dstride; ++j) {
        dot_i += W[di + j] * D[di + j];
    }
    sgPred[i] = sg_d - sqrtf(dot_i);
    
    // compute inner merit, using explore vs exploit coefficient
    innerMerit[i] = muPred[i] - a_d * sgPred[i];

}

void computeInnerEvalations(
    float* V, int Vstride, /* random vecs        */
    float* D, int Dstride, /* distances to known */
    float* W,           /* weights, uses Dstride */
    float* muPred,      /* surrogate expectation */
    float* sgPred,      /* surrogate deviation   */
    float* innerMerit, /* want to minimise this */
    const float sg,     /* kernel deviation      */
    const float l,      /* kernel lengthscale    */
    const float* S,     /* samples, uses Dstride */
    const float* yDiff, /* shared across kernels */
    const float* K, /* inverse covariance matrix */
    const float a,  /* explore vs exploit coeff. */
    const float* lb, const float* ub, /*  bounds */
    int ni          /* number of random vectors  */
) {

    int blockSize = 256;
    int numBlocks = (ni + blockSize - 1) / blockSize;

    hipMemcpyToSymbol(HIP_SYMBOL(sg_d), &sg, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(l_d),  &l,  sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(a_d),  &a,  sizeof(float));

    innerEvaluationsKernel<<<numBlocks, blockSize>>>(
        V, Vstride, D, Dstride, W, muPred, sgPred, innerMerit,
        S, yDiff, K, lb, ub, ni);

    hipDeviceSynchronize();

}




